#include "hip/hip_runtime.h"
// #include "buffer_manager.cuh"
#include <cinttypes>
#include <cstdint>
#include <cstdio>

// #include <cinttypes>

// __device__ __constant__ threadsafe_device_stack<int32_t *, (int32_t *) NULL> * pool;
// __device__ __constant__ int deviceId;
// __device__ __constant__ void * buff_start;
// __device__ __constant__ void * buff_end  ;

#include "common/gpu/gpu-common.hpp"

#include "multigpu/buffer_manager.cuh"
#include "topology/affinity_manager.hpp"
#include "util/raw-memory-manager.hpp"

#include <thread>
// #include <utmpx.h>
// #include <unistd.h>


template<typename T, typename... Args>
__host__ T * cuda_new(int dev, Args... args){
    if (dev >= 0){
        set_device_on_scope d(dev);
        T *tmp = new T(args...);
        T *res;
        gpu_run(hipMalloc((void**) &res, sizeof(T)));
        gpu_run(hipMemcpy(res, tmp, sizeof(T), hipMemcpyDefault));
        gpu_run(hipDeviceSynchronize());
        free(tmp);  //NOTE: bad practice ? we want to allocate tmp by new to
                    //      trigger initialization but we want to free the 
                    //      corresponding memory after moving to device 
                    //      without triggering the destructor
        return res;
    } else {
        T *tmp = new T(args...);
        T *res;
        gpu_run(hipHostMalloc((void**) &res, sizeof(T)));
        gpu_run(hipMemcpy(res, tmp, sizeof(T), hipMemcpyDefault));
        gpu_run(hipDeviceSynchronize());
        free(tmp);  //NOTE: bad practice ? we want to allocate tmp by new to
                    //      trigger initialization but we want to free the 
                    //      corresponding memory after moving to device 
                    //      without triggering the destructor
        return res;
        // return new T(args...);
    }
}


template<typename T, typename... Args>
__host__ void cuda_delete(T *obj, Args... args){
    if (topology::getInstance().getGpuAddressed(obj)){
        T *tmp = (T *) malloc(sizeof(T));
        gpu_run(hipDeviceSynchronize());
        gpu_run(hipMemcpy(tmp, obj, sizeof(T), hipMemcpyDefault));
        gpu_run(hipFree(obj));
        delete tmp;
    } else {
        T *tmp = (T *) malloc(sizeof(T));
        gpu_run(hipDeviceSynchronize());
        gpu_run(hipMemcpy(tmp, obj, sizeof(T), hipMemcpyDefault));
        gpu_run(hipHostFree(obj));
        delete tmp;
        // delete obj;
    }
}





extern "C"{

#ifndef NCUDA
__device__ void dprinti64(int64_t x){
    printf("%" PRId64 "\n", x);
}

__device__ int32_t * get_buffers(){
    uint32_t b = __ballot(1);
    uint32_t m = 1 << get_laneid();
    int32_t * ret;
    do {
        uint32_t leader = b & -b;

        if (leader == m) ret = buffer_manager<int32_t>::get_buffer();

        b ^= leader;
    } while (b);
    return ret;
}

void * get_dev_buffer(){
    return buffer_manager<int32_t>::h_get_buffer(topology::getInstance().getActiveGpu().id);
}

__device__ void release_buffers(int32_t * buff){
    uint32_t b = __ballot(buff != NULL);
    uint32_t m = 1 << get_laneid();
    do {
        uint32_t leader = b & -b;

        if (leader == m) buffer_manager<int32_t>::release_buffer(buff);

        b ^= leader;
    } while (b);
}

__device__ void dprinti(int32_t x){
    printf("%d\n", x);
}

__device__ void dprintptr(void * x){
    printf("%p\n", x);
}
#endif
}

void initializeModule(hipModule_t & cudaModule){
#ifndef NCUDA
    hipDeviceptr_t ptr  ;
    size_t      bytes;
    void *      mem  ;

    gpu_run(hipModuleGetGlobal(&ptr, &bytes, cudaModule, "pool"));
    gpu_run(hipMemcpyFromSymbol(&mem, HIP_SYMBOL(pool      ), sizeof(void   *)));
    gpu_run(hipMemcpyHtoD        (ptr , &mem      , sizeof(void   *)));

    gpu_run(hipModuleGetGlobal(&ptr, &bytes, cudaModule, "buff_start"));
    gpu_run(hipMemcpyFromSymbol(&mem, HIP_SYMBOL(buff_start), sizeof(void   *)));
    gpu_run(hipMemcpyHtoD        (ptr , &mem      , sizeof(void   *)));

    gpu_run(hipModuleGetGlobal(&ptr, &bytes, cudaModule, "buff_end"));
    gpu_run(hipMemcpyFromSymbol(&mem, HIP_SYMBOL(buff_end), sizeof(void   *)));
    gpu_run(hipMemcpyHtoD        (ptr , &mem      , sizeof(void   *)));

    gpu_run(hipModuleGetGlobal(&ptr, &bytes, cudaModule, "deviceId"));
    gpu_run(hipMemcpyFromSymbol(&mem, HIP_SYMBOL(deviceId  ), sizeof(int)));
    gpu_run(hipMemcpyHtoD        (ptr , &mem      , sizeof(int)));
#endif
}


#ifndef NCUDA
__global__ void release_buffer_host(void **buff, int buffs){
    assert(blockDim.x * blockDim.y * blockDim.z == 1);
    assert( gridDim.x *  gridDim.y *  gridDim.z == 1);
    for (int i = 0 ; i < buffs ; ++i) buffer_manager<int32_t>::release_buffer((int32_t *) buff[i]);
}

__global__ void get_buffer_host(void **buff, int buffs){
    assert(blockDim.x * blockDim.y * blockDim.z == 1);
    assert( gridDim.x *  gridDim.y *  gridDim.z == 1);
    for (int i = 0 ; i < buffs ; ++i) buff[i] = buffer_manager<int32_t>::try_get_buffer();
}
#endif

int num_of_gpus;
int num_of_cpus;

inline int get_gpu_count(){
    return  num_of_gpus;
}

inline int get_cpu_numa_node_count(){
    return num_of_cpus;
}

int                                                 cpu_cnt;
// cpu_set_t                                          *gpu_affinity;
cpu_set_t                                          *cpu_numa_affinity;
int                                                *gpu_numa_node;

#if defined(__clang__) && defined(__CUDA__)
template<typename T>
__device__ T * buffer_manager<T>::get_buffer(){
    return pool->pop();
}

template<typename T>
__host__   T * buffer_manager<T>::get_buffer(){
    return get_buffer_numa(get_affinity());
}
#else
template<typename T>
__host__ __device__ T * buffer_manager<T>::get_buffer(){
#ifdef __CUDA_ARCH__
    return pool->pop();
#else
    return get_buffer_numa(get_affinity());
#endif
}
#endif

template<typename T>
__device__ T * buffer_manager<T>::try_get_buffer(){
#ifndef NCUDA
    T * b;
    bool got = pool->pop_if_nonempty(&b);
    if (!got) b = NULL;
    return b;
#else 
    return NULL;
#endif
}

#include <topology/topology.hpp>

template<typename T>
__host__ void buffer_manager<T>::init(int size, int h_size, size_t buff_buffer_size, size_t buff_keep_threshold){
    const topology &topo = topology::getInstance();
    // std::cout << topo << std::endl;

    uint32_t devices = topo.getGpuCount();
    buffer_manager<T>::h_size = h_size;
    
    uint32_t cores   = topo.getCoreCount();

    {

        //FIXME: Generalize
        uint32_t cpu_numa_nodes = topo.getCpuNumaNodeCount();
        uint32_t max_numa_id    = 0;
        for (const auto &n: topo.getCpuNumaNodes()){
            max_numa_id = std::max(max_numa_id, n.id);
        }

        // std::cout << "CPU numa nodes : " << cpu_numa_nodes << std::endl;
        // std::cout << "CPU cores      : " << cores << std::endl;
        // std::cout << "GPU devices    : " << devices << std::endl;

        terminating        = false;
        device_buffs_mutex = new mutex              [devices];
        device_buffs_cv    = new condition_variable [devices];
        device_buffs_thrds = new thread *           [devices];
        device_buffs_pool  = new vector<T *>        [devices];
        release_streams    = new hipStream_t       [devices];

        h_pool             = new h_pool_t *         [cores  ];
        h_pool_numa        = new h_pool_t *         [max_numa_id + 1];

        h_d_pool           = new pool_t            *[devices];

        h_buff_start       = new void              *[devices];
        h_buff_end         = new void              *[devices];

        h_h_buff_start     = new void              *[max_numa_id + 1];

        device_buff        = new T **[devices];
        device_buff_size   = buff_buffer_size;
        keep_threshold     = buff_keep_threshold;

        buffer_cache.clear();

    }

    // gpu_run(hipHostMalloc(&tmp, device_buff_size*sizeof(buffer_t *)*devices));
    // for (int i = 0 ; i < devices ; ++i) {
        // device_buff[i] = tmp + device_buff_size*sizeof(buffer_t *)*i;
    // }

    // gpu_run(nvmlInit());
    // unsigned int device_count = 0;
    // gpu_run(nvmlDeviceGetCount(&device_count));
    // assert(device_count == devices && "NMVL disagrees with cuda about the number of GPUs");


    // gpu_affinity       = new cpu_set_t[devices];
    // cpu_numa_affinity  = new cpu_set_t[cpu_numa_nodes];
    // gpu_numa_node      = new int      [devices];

    // for (int j = 0 ; j < devices        ; ++j) CPU_ZERO(&gpu_affinity[j]);
    // for (int j = 0 ; j < cpu_numa_nodes ; ++j) CPU_ZERO(&cpu_numa_affinity[j]);


    // for (int j = 0 ; j < devices        ; ++j) std::cout << gpu_affinity[j]      << std::endl;
    // for (int j = 0 ; j < cpu_numa_nodes ; ++j) std::cout << cpu_numa_affinity[j] << std::endl;

    // diascld36
    // //FIXME: Generalize
    // for (int i = 0 ; i < 48 ; i += 2) CPU_SET(i, &gpu_affinity[0]);
    // for (int i = 1 ; i < 48 ; i += 2) CPU_SET(i, &gpu_affinity[1]);

    // //FIXME: Generalize
    // cpu_numa_affinity  = new cpu_set_t[cpu_numa_nodes];
    // for (int i = 0 ; i < 48 ; i += 2) CPU_SET(i, &cpu_numa_affinity[0]);
    // for (int i = 1 ; i < 48 ; i += 2) CPU_SET(i, &cpu_numa_affinity[1]);

//     for (int j = 0 ; j < devices ; ++j){
// #ifndef NCUDA
//         int sets = ((cores + 63) / 64);
//         uint64_t cpuSet[sets];
//         for (int i = 0 ; i < sets ; ++i) cpuSet[i] = 0;
//         nvmlDevice_t device;

//         nvmlDeviceGetHandleByIndex(j, &device);
//         nvmlDeviceGetCpuAffinity(device, sets, cpuSet);
//         for (int i = 0 ; i < sets ; ++i){
//             for (int k = 0 ; k < 64 ; ++k){
//                 if ((cpuSet[i] >> k) & 1){
//                     std::cout << "d" << j << " " << (64 * i + k) << std::endl;
//                     CPU_SET(64 * i + k, &gpu_affinity[j]);
//                 }
//             }
//         }
// #else
//         assert(false);
// #endif
//     }

    // for (int j = 0 ; j < devices        ; ++j) std::cout << gpu_affinity[j]      << std::endl;
    // for (int j = 0 ; j < cpu_numa_nodes ; ++j) std::cout << cpu_numa_affinity[j] << std::endl;

    // num_of_gpus = devices;
    // num_of_cpus = 0;

    // std::cout << devices << std::endl;
    // std::cout << cores   << std::endl;

    // for (int j = 0 ; j < cores ; ++j){
    //     std::cout << j << " " << numa_node_of_cpu(j) << std::endl;
    //     CPU_SET(j, &cpu_numa_affinity[numa_node_of_cpu(j)]);
    // }

    // for (int j = 0 ; j < cpu_numa_nodes ; ++j){
    //     std::cout << cpu_numa_affinity[j] << std::endl;
    //     if (CPU_COUNT(&cpu_numa_affinity[j]) > 0) ++num_of_cpus;
    // }



    //numa_node_of_cpu must be set prior to this
    // for (int j = 0 ; j < devices        ; ++j) gpu_numa_node[j] = calc_numa_node_of_gpu(j);

    // for (int i = 0 ; i < cores ; ++i){
    //     std::cout << "CPU " << i << " local to GPU ";
    //     for (int j = 0 ; j < devices ; ++j){
    //         if (CPU_ISSET(i, &gpu_affinity[j])) std::cout << j;
    //     }
    //     std::cout << std::endl;
    // }

    // for (int i = 0 ; i < cores ; ++i){
    //     std::cout << "CPU " << i << " local to NUMA ";
    //     for (int j = 0 ; j < cpu_numa_nodes ; ++j){
    //         if (CPU_ISSET(i, &cpu_numa_affinity[j])) std::cout << j;
    //     }
    //     std::cout << std::endl;
    // }


    // // diascld37
    // //FIXME: Generalize
    // for (int i = 0  ; i < 14 ; ++i) CPU_SET(i, &gpu_affinity[0]);
    // for (int i = 0  ; i < 14 ; ++i) CPU_SET(i, &gpu_affinity[1]);
    // for (int i = 14 ; i < 28 ; ++i) CPU_SET(i, &gpu_affinity[2]);
    // for (int i = 14 ; i < 28 ; ++i) CPU_SET(i, &gpu_affinity[3]);

    //FIXME: Generalize
    // cpu_numa_affinity  = new cpu_set_t[cpu_numa_nodes];
    // for (int i = 0  ; i < 14 ; ++i) CPU_SET(i, &cpu_numa_affinity[0]);
    // for (int i = 14 ; i < 28 ; ++i) CPU_SET(i, &cpu_numa_affinity[1]);

    mutex buff_cache;

    vector<thread> buffer_pool_constrs;
    for (const auto &gpu: topo.getGpus()){
        buffer_pool_constrs.emplace_back([gpu, size, &buff_cache]{
            uint32_t j = gpu.id;

            set_exec_location_on_scope d(j);

            T      *mem;
            size_t  pitch;
            gpu_run(hipMallocPitch(&mem, &pitch, h_vector_size*sizeof(T), size));
            
            vector<T *> buffs;
            
            buffs.reserve(size);
            for (size_t i = 0 ; i < size ; ++i) {
                T        * m = (T *) (((char *) mem) + i*pitch);
                // buffer_t * b = cuda_new<buffer_t>(j, m, j);
                buffs.push_back(m);

                // cout << "Device " << j << " : data = " << m << endl;
                assert(topology::getInstance().getGpuAddressed(m)->id == j);
            }
            {   //FIXME: why are we including device buffers in the cache?
                lock_guard<mutex> guard(buff_cache);
                for (const auto b: buffs) buffer_cache[b] = 0;
            }
            
            pool_t * tmp =  cuda_new<pool_t>(j, size, buffs, j);
            gpu_run(hipMemcpyToSymbol(HIP_SYMBOL(pool      ), &tmp, sizeof(pool_t *)));
            gpu_run(hipMemcpyToSymbol(HIP_SYMBOL(deviceId  ),   &j, sizeof(int     )));
            gpu_run(hipMemcpyToSymbol(HIP_SYMBOL(buff_start), &mem, sizeof(void   *)));
            void * e = (void *) (((char *) mem) + size*pitch);
            gpu_run(hipMemcpyToSymbol(HIP_SYMBOL(buff_end  ),   &e, sizeof(void   *)));

            h_d_pool    [j] = tmp;
            h_buff_start[j] = mem;
            h_buff_end  [j] = e  ;
            
            int greatest;
            int lowest;
            gpu_run(hipDeviceGetStreamPriorityRange(&greatest, &lowest));
            // std::cout << greatest << " " << lowest << std::endl;
            gpu_run(hipStreamCreateWithPriority(&(release_streams[j]), hipStreamNonBlocking, lowest));

            T **bf;
            gpu_run(hipHostMalloc(&bf, std::max(device_buff_size, keep_threshold)*sizeof(T *)));
            device_buff[j] = bf;

            device_buffs_thrds[j] = new thread(dev_buff_manager, j);
        });
    }


    for (const auto &cpu: topo.getCpuNumaNodes()){
        buffer_pool_constrs.emplace_back([cpu, h_size, cores, &buff_cache]{
            set_exec_location_on_scope cu{cpu};
            const auto &topo = topology::getInstance();

            size_t bytes = h_vector_size * sizeof(T) * h_size;
            T      *mem = (T *) RawMemoryManager::mallocPinned(bytes);
            assert(mem);

            // T * mem;
            // gpu_run(hipHostMalloc(&mem, h_vector_size*sizeof(T)*h_size));
            printf("Memory at %p is at node %d (expected: %d)\n", mem, topo.getCpuNumaNodeAddressed(mem)->id, get_affinity().id);
            // assert(topo.getCpuNumaNodeAddressed(mem)->id == cpu.id); //FIXME: fails on power9, should reenable after we fix it

            h_h_buff_start[cpu.id] = mem;

            vector<T *> buffs;
            buffs.reserve(h_size);
            for (size_t j = 0 ; j < h_size ; ++j) {
                T        * m = mem + j * h_vector_size;
                // buffer_t * b = cuda_new<buffer_t>(-1, m, -1);
                buffs.push_back(m);

                m[0] = 0; //force allocation of first page of each buffer
                // cout << "NUMA " << topo.getCpuNumaNodeAddressed(m)->id << " : data = " << m << endl;
                // assert(topo.getCpuNumaNodeAddressed(m)->id == cpu.id); //FIXME: fails on power9, should reenable after we fix it
            }

            {
                lock_guard<mutex> guard(buff_cache);
                for (const auto b: buffs) buffer_cache[b] = 0;
            }

            h_pool_t *p         = new h_pool_t(h_size, buffs);
            
            h_pool_numa[cpu.id] = p;

            for (const auto &core: cpu.local_cores) h_pool[core] = p;
        });
    }

    // h_pool_t **numa_h_pools = new h_pool_t *[cpu_numa_nodes];

    // for (int i = 0 ; i < cores ; ++i) numa_node_inited[i] = NULL;

    // for (int i = 0 ; i < cores ; ++i){
    //     int numa_node = numa_node_of_cpu(i);

    //     if (!numa_node_inited[numa_node]){
    //         cpu_set_t cpuset;
    //         CPU_ZERO(&cpuset);
    //         CPU_SET(i, &cpuset);


    //         T      *mem;
    //         gpu_run(hipHostMalloc(&mem, buffer_t::capacity()*sizeof(T)*size));

    //         vector<buffer_t *> buffs;
    //         for (size_t i = 0 ; i < size ; ++i) buffs.push_back(cuda_new<buffer_t>(-1, mem + i * buffer_t::capacity(), -1));
    //         numa_node_inited[numa_node] = new h_pool_t(size, buffs);
    //     }
    //     h_pool[i] = numa_node_inited[numa_node];
    // }

    // T      *mem;
    // gpu_run(hipHostMalloc(&mem, buffer_t::capacity()*sizeof(T)*size));

    // vector<buffer_t *> buffs;
    // for (size_t i = 0 ; i < size ; ++i) buffs.push_back(cuda_new<buffer_t>(-1, mem + i * buffer_t::capacity(), -1));
    // h_pool = new h_pool_t(size, buffs);

    for (auto &t: buffer_pool_constrs) t.join();

    buffer_logger = new thread{buffer_manager<T>::log_buffers};
}

template<typename T>
__host__ void buffer_manager<T>::destroy(){
    int devices;
    gpu_run(hipGetDeviceCount(&devices));

    // long cores = sysconf(_SC_NPROCESSORS_ONLN);
    // assert(cores > 0);

    // int cpu_numa_nodes = numa_num_task_nodes();

    terminating = true;

    buffer_logger->join();
    // device_buffs_mutex = new mutex              [devices];
    // device_buffs_pool  = new vector<buffer_t *> [devices];
    // release_streams    = new hipStream_t       [devices];

    // h_pool             = new h_pool_t *         [cores  ];
    // h_pool_numa        = new h_pool_t *         [cpu_numa_nodes];

    // device_buff        = new buffer_t**[devices];
    // device_buff_size   = buff_buffer_size;
    // keep_threshold     = buff_keep_threshold;

    // gpu_affinity       = new cpu_set_t[devices];

    // mutex buff_cache;

    const auto &topo = topology::getInstance();

    vector<thread> buffer_pool_constrs;
    for (const auto &gpu: topo.getGpus()) {
#ifndef NCUDA
        buffer_pool_constrs.emplace_back([gpu]{
            uint32_t j = gpu.id;
            set_device_on_scope d(j);

            device_buffs_cv[j].notify_all();
            device_buffs_thrds[j]->join();

            std::unique_lock<std::mutex> lock(device_buffs_mutex[j]);

            size_t size = device_buffs_pool[j].size();
            assert(size <= keep_threshold);
            for (size_t i = 0 ; i < size ; ++i) device_buff[j][i] = device_buffs_pool[j][i];

            release_buffer_host<<<1, 1, 0, release_streams[j]>>>((void **) device_buff[j], size);
            gpu_run(hipStreamSynchronize(release_streams[j]));

            pool_t *tmp;
            gpu_run(hipMemcpyFromSymbol(&tmp, HIP_SYMBOL(pool), sizeof(pool_t *)));
            cuda_delete(tmp);

            T * mem;
            gpu_run(hipMemcpyFromSymbol(&mem, HIP_SYMBOL(buff_start), sizeof(void   *)));
            gpu_run(hipFree(mem));

            gpu_run(hipStreamDestroy(release_streams[j]));

            gpu_run(hipHostFree(device_buff[j]));
        });
#else
        assert(false);
#endif
    }
    
    size_t h_size = buffer_manager<T>::h_size;

    for (const auto &cpu: topo.getCpuNumaNodes()){
        buffer_pool_constrs.emplace_back([cpu, h_size]{
            set_exec_location_on_scope cu{cpu};
            RawMemoryManager::freePinned(h_h_buff_start[cpu.id]);
            delete h_pool_numa[cpu.id];
        });
    }

    for (auto &t: buffer_pool_constrs) t.join();

    terminating        = false;
    delete[] device_buffs_mutex;
    delete[] device_buffs_cv   ;
    delete[] device_buffs_thrds;
    delete[] device_buffs_pool ;
    delete[] release_streams   ;

    delete[] h_pool            ;
    delete[] h_pool_numa       ;

    delete[] h_d_pool          ;

    delete[] h_buff_start      ;
    delete[] h_buff_end        ;

    delete[] h_h_buff_start    ;

    delete[] device_buff       ;

    buffer_cache.clear();
}

extern "C"{
    void * get_buffer(size_t bytes){
        assert(bytes <= sizeof(int32_t) * h_vector_size); //FIMXE: buffer manager should be able to allocate blocks of arbitary size
        return (void *) buffer_manager<int32_t>::h_get_buffer(-1);
    }

    void   release_buffer(void * buff){
        buffer_manager<int32_t>::release_buffer((int32_t *) buff);
    }
}

template<typename T>
void buffer_manager<T>::dev_buff_manager(int dev){
#ifndef NCUDA
    set_device_on_scope d(dev);

    while (true){
        bool sleep = false;
        int added = 0;
        {
            std::unique_lock<mutex> lk(device_buffs_mutex[dev]);
            
            device_buffs_cv[dev].wait(lk, [dev]{return device_buffs_pool[dev].empty() || terminating;});

            if (terminating) break;

            get_buffer_host<<<1, 1, 0, release_streams[dev]>>>((void **) device_buff[dev], device_buff_size);
            gpu_run(hipStreamSynchronize(release_streams[dev]));


            for (size_t i = 0 ; i < device_buff_size ; ++i){
                if (device_buff[dev][i]) {
                    device_buffs_pool[dev].push_back(device_buff[dev][i]);
                    ++added;
                }
            }

            device_buffs_cv[dev].notify_all();

            sleep = device_buffs_pool[dev].empty();

            lk.unlock();
        }

        if (sleep) {
            std::cout << "Sleeping... (" << added << ")" << std::endl;
            std::this_thread::sleep_for(std::chrono::seconds(1));
            std::cout << "Waking..." << std::endl;
        }
        // device_buffs_pool[dev].insert(device_buffs_pool[dev].end(), device_buff[dev], device_buff[dev]+device_buff_size);

        // lk.unlock();
    }
#endif
}
 
template<typename T>
__host__ void buffer_manager<T>::log_buffers(){
    return;
    const auto &topo = topology::getInstance();
    uint32_t devices = topo.getGpuCount();
    if (devices <= 0) return;

    uint32_t        cnts[devices];
    hipStream_t    strs[devices];
    
    for (const auto &gpu: topo.getGpus()){
        set_device_on_scope d(gpu);
        gpu_run(hipStreamCreateWithFlags(strs+gpu.id, hipStreamNonBlocking));
    }

    char progress[]{"-\\|/"};
    size_t iter = 0;

    while (!terminating){
        std::this_thread::sleep_for(std::chrono::milliseconds(250));
        for (uint32_t i = 0 ; i < devices ; ++i){
            gpu_run(hipMemcpyAsync(cnts + i, (void *) &(h_d_pool[i]->cnt), sizeof(decltype(pool_t::cnt)), hipMemcpyDefault, strs[i]));
        }
        for (uint32_t i = 0 ; i < devices ; ++i) gpu_run(hipStreamSynchronize(strs[i]));
        std::cerr << "\0337\033[H\r";
        for (uint32_t i = 0 ; i < 80 ; ++i) std::cerr << ' ';
        std::cerr << "\rBuffers on device: ";
        for (uint32_t i = 0 ; i < devices ; ++i) std::cerr << cnts[i] << "(+" << device_buffs_pool[i].size() << ") ";
        std::cerr << "\t\t" << progress[(iter++) % (sizeof(progress) - 1)]; //for null character
        std::cerr << "\0338";
        std::cerr.flush();
    }

    for (const auto &gpu: topo.getGpus()){
        set_device_on_scope d(gpu);
        gpu_run(hipStreamDestroy(strs[gpu.id]));
    }
}

template<typename T>
__host__ inline T * buffer_manager<T>::h_get_buffer(int dev){
    if (dev >= 0){
        std::unique_lock<std::mutex> lock(device_buffs_mutex[dev]);

        device_buffs_cv[dev].wait(lock, [dev]{return !device_buffs_pool[dev].empty();});

        T * ret = device_buffs_pool[dev].back();
        device_buffs_pool[dev].pop_back();
        device_buffs_cv[dev].notify_all();
        return ret;
    } else {
        return get_buffer();
    }
}


template<typename T>
__host__ void buffer_manager<T>::overwrite(T * buff, const T * data, uint32_t N, hipStream_t strm, bool blocking){
#ifndef NCUDA
    gpu_run(hipMemcpyAsync(buff, data, N*sizeof(T), hipMemcpyDefault, strm));
    if (blocking) gpu_run(hipStreamSynchronize(strm));
#else
    memcpy(buff, data, N*sizeof(T));
#endif
}

template<typename T>
__host__ void buffer_manager<T>::overwrite_bytes(void * buff, const void * data, size_t bytes, hipStream_t strm, bool blocking){
#ifndef NCUDA
    gpu_run(hipMemcpyAsync(buff, data, bytes, hipMemcpyDefault, strm));
    if (blocking) gpu_run(hipStreamSynchronize(strm));
#else
    memcpy(buff, data, bytes);
#endif
}

template class buffer_manager<int32_t>;

#ifndef NCUDA
__global__ void GpuHashRearrange_acq_buffs(void   ** buffs){
    buffs[blockIdx.x] = get_buffers();
}
#endif

extern "C"{
void gpu_memset(void * dst, int32_t val, size_t size){
    hipStream_t strm;
    gpu_run(hipStreamCreateWithFlags(&strm, hipStreamNonBlocking));
    gpu_run(hipMemsetAsync(dst, val, size, strm));
    gpu_run(hipStreamSynchronize(strm));
    gpu_run(hipStreamDestroy(strm));
}
}
